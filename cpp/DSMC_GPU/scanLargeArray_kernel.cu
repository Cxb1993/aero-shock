#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// includes, kernels
#include <hip/hip_runtime.h>
#include <assert.h>

#define float int

#ifndef _SCAN_BEST_KERNEL_CU_
#define _SCAN_BEST_KERNEL_CU_

// Define this to more rigorously avoid bank conflicts, 
// even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance 
// is lower with ZERO_BANK_CONFLICTS enabled.  It is provided
// as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www.cs.cmu.edu/~blelloch/papers/Ble93.pdf
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void loadSharedChunkFromMem(float *s_data,
                                       const float *g_idata, 
                                       int n, int baseIndex,
                                       int& ai, int& bi, 
                                       int& mem_ai, int& mem_bi, 
                                       int& bankOffsetA, int& bankOffsetB)
{
    int thid = threadIdx.x;
    mem_ai = baseIndex + threadIdx.x;
    mem_bi = mem_ai + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

    // compute spacing to avoid bank conflicts
    bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    // pad values beyond n with zeros
    s_data[ai + bankOffsetA] = g_idata[mem_ai]; 
    
    if (isNP2) // compile-time decision
    {
        s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0; 
    }
    else
    {
        s_data[bi + bankOffsetB] = g_idata[mem_bi]; 
    }
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(float* g_odata, 
                                      const float* s_data,
                                      int n, 
                                      int ai, int bi, 
                                      int mem_ai, int mem_bi,
                                      int bankOffsetA, int bankOffsetB)
{
    __syncthreads();

    // write results to global memory
    g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
    if (isNP2) // compile-time decision
    {
        if (bi < n)
            g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
    else
    {
        g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
}

template <bool storeSum>
__device__ void clearLastElement(float* s_data, 
                                 float *g_blockSums, 
                                 int blockIndex)
{
    if (threadIdx.x == 0)
    {
        int index = (blockDim.x << 1) - 1;
        index += CONFLICT_FREE_OFFSET(index);
        
        if (storeSum) // compile-time decision
        {
            // write this block's total sum to the corresponding index in the blockSums array
            g_blockSums[blockIndex] = s_data[index];
        }

        // zero the last element in the scan so it will propagate back to the front
        s_data[index] = 0;
    }
}



__device__ unsigned int buildSum(float *s_data)
{
    unsigned int thid = threadIdx.x;
    unsigned int stride = 1;
    
    // build the sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    return stride;
}

__device__ void scanRootToLeaves(float *s_data, unsigned int stride)
{
     unsigned int thid = threadIdx.x;

    // traverse down the tree building the scan in place
    for (int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
}

template <bool storeSum>
__device__ void prescanBlock(float *data, int blockIndex, float *blockSums)
{
    int stride = buildSum(data);               // build the sum in place up the tree
    clearLastElement<storeSum>(data, blockSums, 
                               (blockIndex == 0) ? blockIdx.x : blockIndex);
    scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

template <bool storeSum, bool isNP2>
__global__ void prescan(float *g_odata, 
                        const float *g_idata, 
                        float *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex)
{
    int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
    extern __shared__ float s_data[];

    // load data into shared memory
    loadSharedChunkFromMem<isNP2>(s_data, g_idata, n, 
                                  (baseIndex == 0) ? 
                                  __mul24(blockIdx.x, (blockDim.x << 1)):baseIndex,
                                  ai, bi, mem_ai, mem_bi, 
                                  bankOffsetA, bankOffsetB); 
    // scan the data in each block
    prescanBlock<storeSum>(s_data, blockIndex, g_blockSums); 
    // write results to device memory
    storeSharedChunkToMem<isNP2>(g_odata, s_data, n, 
                                 ai, bi, mem_ai, mem_bi, 
                                 bankOffsetA, bankOffsetB);  
}


__global__ void uniformAdd(float *g_data, 
                           float *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
    g_data[address]              += uni;
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}


#endif // #ifndef _SCAN_BEST_KERNEL_CU_

#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

inline bool 
isPowerOfTwo(int n)
{
    return ((n&(n-1))==0) ;
}

inline int 
floorPow2(int n)
{
#ifdef WIN32
    // method 2
    return 1 << (int)logb((float)n);
#else
    // method 1
    // float nf = (float)n;
    // return 1 << (((*(int*)&nf) >> 23) - 127); 
    int exp;
    frexp((float)n, &exp);
    return 1 << (exp - 1);
#endif
}

//#define BLOCK_SIZE 256
//By CC
#define BLOCK_SIZE 512

float** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

extern "C"
void preallocBlockSums(unsigned int maxNumElements)
{
    assert(g_numEltsAllocated == 0); // shouldn't be called 

    g_numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;

    int level = 0;

    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (float**) malloc(level * sizeof(float*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) 
        {
            hipMalloc((void**) &g_scanBlockSums[level++],  
                                numBlocks * sizeof(float));
        }
        numElts = numBlocks;
    } while (numElts > 1);

    //cutilCheckMsg("preallocBlockSums");
}

extern "C"
void deallocBlockSums()
{
    for (unsigned int i = 0; i < g_numLevelsAllocated; i++)
    {
        hipFree(g_scanBlockSums[i]);
    }

    //cutilCheckMsg("deallocBlockSums");
    
    free((void**)g_scanBlockSums);

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}


void prescanArrayRecursive(float *outArray, 
                           const float *inArray, 
                           int numElements, 
                           int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = 
        max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = 
        numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock)
    {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);    
        
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = 
            sizeof(float) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = 
        sizeof(float) * (numEltsPerBlock + extraSpace);

#ifdef DEBUG
    if (numBlocks > 1)
    {
        assert(g_numEltsAllocated >= numElements);
    }
#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // make sure there are no CUDA errors before we start
    //cutilCheckMsg("prescanArrayRecursive before kernels");

    // execute the scan
    if (numBlocks > 1)
    {
        prescan<true, false><<< grid, threads, sharedMemSize >>>(outArray, 
                                                                 inArray, 
                                                                 g_scanBlockSums[level],
                                                                 numThreads * 2, 0, 0);
        //cutilCheckMsg("prescanWithBlockSums");
        if (np2LastBlock)
        {
            prescan<true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>>
                (outArray, inArray, g_scanBlockSums[level], numEltsLastBlock, 
                 numBlocks - 1, numElements - numEltsLastBlock);
            //cutilCheckMsg("prescanNP2WithBlockSums");
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be sdded to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(g_scanBlockSums[level], 
                              g_scanBlockSums[level], 
                              numBlocks, 
                              level+1);

        uniformAdd<<< grid, threads >>>(outArray, 
                                        g_scanBlockSums[level], 
                                        numElements - numEltsLastBlock, 
                                        0, 0);
        //cutilCheckMsg("uniformAdd");
        if (np2LastBlock)
        {
            uniformAdd<<< 1, numThreadsLastBlock >>>(outArray, 
                                                     g_scanBlockSums[level], 
                                                     numEltsLastBlock, 
                                                     numBlocks - 1, 
                                                     numElements - numEltsLastBlock);
            //cutilCheckMsg("uniformAdd");
        }
    }
    else if (isPowerOfTwo(numElements))
    {
        prescan<false, false><<< grid, threads, sharedMemSize >>>(outArray, inArray,
                                                                  0, numThreads * 2, 0, 0);
        //cutilCheckMsg("prescan");
    }
    else
    {
         prescan<false, true><<< grid, threads, sharedMemSize >>>(outArray, inArray, 
                                                                  0, numElements, 0, 0);
         //cutilCheckMsg("prescanNP2");
    }
}

extern "C"
void prescanArray(float *outArray, float *inArray, int numElements)
{
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}


#endif // _PRESCAN_CU_


/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#ifndef _SCAN_BEST_KERNEL_CU_
#define _SCAN_BEST_KERNEL_CU_

// Define this to more rigorously avoid bank conflicts, 
// even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance 
// is lower with ZERO_BANK_CONFLICTS enabled.  It is provided
// as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www.cs.cmu.edu/~blelloch/papers/Ble93.pdf
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void loadSharedChunkFromMem(float *s_data,
                                       const float *g_idata, 
                                       int n, int baseIndex,
                                       int& ai, int& bi, 
                                       int& mem_ai, int& mem_bi, 
                                       int& bankOffsetA, int& bankOffsetB)
{
    int thid = threadIdx.x;
    mem_ai = baseIndex + threadIdx.x;
    mem_bi = mem_ai + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

    // compute spacing to avoid bank conflicts
    bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    // pad values beyond n with zeros
    s_data[ai + bankOffsetA] = g_idata[mem_ai]; 
    
    if (isNP2) // compile-time decision
    {
        s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0; 
    }
    else
    {
        s_data[bi + bankOffsetB] = g_idata[mem_bi]; 
    }
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(float* g_odata, 
                                      const float* s_data,
                                      int n, 
                                      int ai, int bi, 
                                      int mem_ai, int mem_bi,
                                      int bankOffsetA, int bankOffsetB)
{
    __syncthreads();

    // write results to global memory
    g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
    if (isNP2) // compile-time decision
    {
        if (bi < n)
            g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
    else
    {
        g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
}

template <bool storeSum>
__device__ void clearLastElement(float* s_data, 
                                 float *g_blockSums, 
                                 int blockIndex)
{
    if (threadIdx.x == 0)
    {
        int index = (blockDim.x << 1) - 1;
        index += CONFLICT_FREE_OFFSET(index);
        
        if (storeSum) // compile-time decision
        {
            // write this block's total sum to the corresponding index in the blockSums array
            g_blockSums[blockIndex] = s_data[index];
        }

        // zero the last element in the scan so it will propagate back to the front
        s_data[index] = 0;
    }
}



__device__ unsigned int buildSum(float *s_data)
{
    unsigned int thid = threadIdx.x;
    unsigned int stride = 1;
    
    // build the sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    return stride;
}

__device__ void scanRootToLeaves(float *s_data, unsigned int stride)
{
     unsigned int thid = threadIdx.x;

    // traverse down the tree building the scan in place
    for (int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
}

template <bool storeSum>
__device__ void prescanBlock(float *data, int blockIndex, float *blockSums)
{
    int stride = buildSum(data);               // build the sum in place up the tree
    clearLastElement<storeSum>(data, blockSums, 
                               (blockIndex == 0) ? blockIdx.x : blockIndex);
    scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

template <bool storeSum, bool isNP2>
__global__ void prescan(float *g_odata, 
                        const float *g_idata, 
                        float *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex)
{
    int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
    extern __shared__ float s_data[];

    // load data into shared memory
    loadSharedChunkFromMem<isNP2>(s_data, g_idata, n, 
                                  (baseIndex == 0) ? 
                                  __mul24(blockIdx.x, (blockDim.x << 1)):baseIndex,
                                  ai, bi, mem_ai, mem_bi, 
                                  bankOffsetA, bankOffsetB); 
    // scan the data in each block
    prescanBlock<storeSum>(s_data, blockIndex, g_blockSums); 
    // write results to device memory
    storeSharedChunkToMem<isNP2>(g_odata, s_data, n, 
                                 ai, bi, mem_ai, mem_bi, 
                                 bankOffsetA, bankOffsetB);  
}


__global__ void uniformAdd(float *g_data, 
                           float *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
    g_data[address]              += uni;
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}


#endif // #ifndef _SCAN_BEST_KERNEL_CU_
